#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "hip/hip_runtime_api.h"

typedef unsigned int uint;
typedef unsigned long ulong;

extern "C"
{
    void cuda_init(int argc, const char **argv)
    {
        int devID;

        // use device with highest Gflops/s
        devID = findCudaDevice(argc, argv);

        if (devID < 0)
        {
            printf("No CUDA Capable devices found, exiting...\n");
            exit(EXIT_SUCCESS);
        }
    }
    void cuda_destroy()
    {
        // hipDeviceReset causes the driver to clean up all state. While
        // not mandatory in normal operation, it is good practice.  It is also
        // needed to ensure correct operation when the application is being
        // profiled. Calling hipDeviceReset causes all profile data to be
        // flushed before the application exits
        hipDeviceReset();
    }

    void cuda_allocateArray(void **devPtr, size_t size)
    {
        checkCudaErrors(hipMalloc(devPtr, size));
    }

    void cuda_freeArray(void *devPtr)
    {
        checkCudaErrors(hipFree(devPtr));
    }

    void cuda_copyArrayToDevice(void *device, const void *host, int offset, int size)
    {
        checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
    }

    void cuda_copyArrayFromDevice(void *host, const void *device, int size)
    {
        checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
    }

    void cuda_registerGLTexture(hipGraphicsResource_t* resource, GLuint tex, GLenum target, hipGraphicsRegisterFlags flags)
    {
        checkCudaErrors(hipGraphicsGLRegisterImage(resource, tex, target, flags));
    }

    void cuda_unregisterResource(hipGraphicsResource_t resource)
    {
        checkCudaErrors(hipGraphicsUnregisterResource(resource));
    }

    void cuda_graphicsMapResource(hipGraphicsResource_t *res)
    {
        checkCudaErrors(hipGraphicsMapResources(1, res));
    }

    void cuda_graphicsUnmapResource(hipGraphicsResource_t *res)
    {
        checkCudaErrors(hipGraphicsUnmapResources(1, res));
    }

    void cuda_graphicsSubResourceGetMappedArray(hipArray_t *array, hipGraphicsResource_t res, GLuint index, GLuint level)
    {
        checkCudaErrors(hipGraphicsSubResourceGetMappedArray(array, res, index, level));
    }

    void cuda_createSurfaceObject(hipSurfaceObject_t *surface, hipResourceDesc *desc)
    {
        checkCudaErrors(hipCreateSurfaceObject(surface, desc));
    }

    void cuda_destroySurfaceObject(hipSurfaceObject_t surface)
    {
        checkCudaErrors(hipDestroySurfaceObject(surface));
    }

    void cuda_streamSynchronize(hipStream_t stream)
    {
        checkCudaErrors(hipStreamSynchronize(stream));
    }

    void cuda_deviceSynchronize()
    {
        checkCudaErrors(hipDeviceSynchronize());
    }

    //Round a / b to nearest higher integer value
    uint iDivUp(uint a, uint b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }

    // compute grid and thread block size for a given number of elements
    void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
    {
        numThreads = min(blockSize, n);
        numBlocks = iDivUp(n, numThreads);
    }
}
