#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include "hip/hip_runtime_api.h"

typedef unsigned int uint;
typedef unsigned long ulong;

extern "C"
{
    void cudaInit(int argc, const char **argv)
    {
        int devID;

        // use device with highest Gflops/s
        devID = findCudaDevice(argc, argv);

        if (devID < 0)
        {
            printf("No CUDA Capable devices found, exiting...\n");
            exit(EXIT_SUCCESS);
        }
    }
    void cudaDestroy()
    {
        // hipDeviceReset causes the driver to clean up all state. While
        // not mandatory in normal operation, it is good practice.  It is also
        // needed to ensure correct operation when the application is being
        // profiled. Calling hipDeviceReset causes all profile data to be
        // flushed before the application exits
        hipDeviceReset();
    }

    void allocateArray(void **devPtr, size_t size)
    {
        checkCudaErrors(hipMalloc(devPtr, size));
    }

    void freeArray(void *devPtr)
    {
        checkCudaErrors(hipFree(devPtr));
    }

    void copyArrayToDevice(void *device, const void *host, int offset, int size)
    {
        checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
    }

    void copyArrayFromDevice(void *host, const void *device, int size)
    {
        checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
    }

    ulong sumNumbers(ulong *dNumbers, ulong n)
    {
        // simple reduction from 1 to n
        thrust::device_ptr<ulong> dp_numbers(dNumbers);
        return thrust::reduce(dp_numbers, dp_numbers + n);
        // return 7;
    }

    //Round a / b to nearest higher integer value
    uint iDivUp(uint a, uint b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }

    // compute grid and thread block size for a given number of elements
    void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
    {
        numThreads = min(blockSize, n);
        numBlocks = iDivUp(n, numThreads);
    }
}
