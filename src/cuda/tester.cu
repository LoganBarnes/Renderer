#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "helper_grid.h"
#include "random_kernel.cu"
#include "renderObjects.hpp"

extern "C"
{
    __global__
    void testSamplePoint_kernel(hiprandState *state, Shape *shape, float3 *results, uint numResults, bool useNormals)
    {
        uint idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < numResults)
        {
            SurfaceElement surfel = samplePoint(state, idx, *shape);
            if (useNormals)
                results[idx] = surfel.normal;
            else
                results[idx] = surfel.point;
        }
    }

    void cuda_testSamplePoint(hiprandState *state, Shape *shape, float3 *results, uint numResults, bool useNormals)
    {
        dim3 thread(64);
        dim3 block(1);
        computeGridSize(numResults, thread.x, block.x, thread.x);

        testSamplePoint_kernel<<<block, thread>>>(state, shape, results, numResults, useNormals);
    }

}
