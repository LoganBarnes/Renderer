#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hip/hip_runtime_api.h" // includes hip/hip_vector_types.h
#include "renderObjects.hpp"
#include "renderTypes.hpp"

__constant__ float INF = 1000.0f;
__constant__ float EPS = 1e-7f;

extern "C"
{

    __device__
    int solveQuadratic(float a, float b, float c, float *t1, float *t2)
    {
        float discriminant = b * b - 4.f * a * c;

        // Discriminant is 0. One solution exists.
        if (abs(discriminant) < EPS) // epsilon value
        {
//            if (abs(b) < EPS)
//                return 0;

//            if (abs(a) < EPS)
//            {
//                return 0;
//            }

            *t1 = -b / (2.0 * a);
            return 1;
        }

        // Discriminant is less than 0. No solutions exists.
        if (discriminant < 0.0)
            return 0;

        // Discriminant is greater than 0. Two solutions exists.
//         if (abs(a) < EPS)
//         {
//            return 0;
//         }
         else
        {
            float sqrtDisc = sqrt(discriminant);
            *t1 = (-b + sqrtDisc) / (2.0 * a);
            *t2 = (-b - sqrtDisc) / (2.0 * a);
        }
        return 2;
    }

    __device__
    float4 intersectSphere(float3 E, float3 D)
    {
        float4 n = make_float4(0, 0, 0, INF);
        float3 p;

        float a = dot(D, D);
        float b = 2.0 * dot(D, E);
        float c = dot(E, E) - 1.0;

        float t1, t2;
        int solutions = solveQuadratic(a, b, c, &t1, &t2);

        if (solutions > 0)
        {
            if (t1 > EPS && t1 < n.w)
            {
                p = E + t1 * D;
                n = make_float4(p, t1);
            }
            if (solutions > 1 && t2 < n.w && t2 > EPS)
            {
                p = E + t2 * D;
                n = make_float4(p, t2);
            }
        }

        return n;
    }



    __device__
    float4 intersectQuad(float3 E, float3 D)
    {
        if (D.z < 0.f)
            return make_float4(0.f, 0.f, 0.f, INF);

        /*
         * norm = (0, 0, -1)
         * t = dot((0, 0, 0) - E), norm) / dot(D, norm)
         */
        float t = E.z / (-D.z);
        float3 p = E + t * D;
        if (t < INF && t > 0.f)
        {
            if (p.x >= -1.f && p.x <= 1.f && p.y >= -1.f && p.y <= 1.f)
                return make_float4(0.f, 0.f, -1.f, t);
        }

        return make_float4(0.f, 0.f, 0.f, INF);
    }

    // check for intersections with every shape except the excluded one
    __device__
    bool intersectWorld(Ray *r, Shape *shapes, uint numShapes, SurfaceElement *surfel, int exclude)
    {
        float4 n = make_float4(INF);
        float4 tempN = make_float4(INF);
        Shape s;
        surfel->index = -1;

        for (int i = 0; i < MAX_SHAPES; ++i)
        {
            if (i >= numShapes)
                break;

            if (i == exclude)
                continue;

            Shape &shape = shapes[i];

            float3 E = make_float3(shape.inv * make_float4(r->orig, 1.0));
            float3 D = make_float3(shape.inv * make_float4(r->dir, 0.0));

            // check bounding box first
//            if (!intersectCubeQuick(E, D, INF))
//                continue;

            switch (shape.type)
            {
//            case CONE:
//                tempN = intersectCone(E, D);
//                if (tempN.w < n.w)
//                {
//                    n = tempN;
//                    index = i;
//                    s = shape;
//                }
//                break;
//            case CUBE:
//                tempN = intersectCube(E, D);
//                if (tempN.w < n.w)
//                {
//                    n = tempN;
//                    index = i;
//                    s = shape;
//                }
//                break;
//            case CYLINDER:
//                tempN = intersectCylinder(E, D);
//                if (tempN.w < n.w)
//                {
//                    n = tempN;
//                    index = i;
//                    s = shape;
//                }
//                break;
            case SPHERE:
                tempN = intersectSphere(E, D);
                if (tempN.w < n.w)
                {
                    n = tempN;
                    surfel->index = i;
                    s = shape;
                }
                break;
            case QUAD:
                tempN = intersectQuad(E, D);
                if (tempN.w < n.w)
                {
                    n = tempN;
                    surfel->index = i;
                    s = shape;
                }
                break;
            default:
                break;
            }
        }


        if (surfel->index >= 0)
        {
            surfel->point = r->orig + r->dir * n.w;
            surfel->normal = normalize(s.normInv * normalize(make_float3(n)));

            surfel->material = s.material;

            return true;
        }

        return false;
    }
}
